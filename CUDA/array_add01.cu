#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nvToolsExt.h"

void add_cpu(int* array_a, int* array_b, int* array_c, int elements) { // Dodawanie na CPU
    nvtxRangePushA("add_cpu"); // Otwiera zakres czasu, który potem będzie widoczny w profilerze
    for(int i = 0; i < elements; ++i)
        array_c[i] = array_a[i] + array_b[i];
    nvtxRangePop(); // Zamyka powyższy zakres
}

__global__ void add_gpu(int* array_a, int* array_b, int* array_c) { // Dodawanie na GPU
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    array_c[index] = array_a[index] + array_b[index];
}

int main() {
    int elements = 1 << 28; // Rozmiar tablic

    int* host_a = (int*) malloc(sizeof(int) * elements); // Tablice z danymi po stronie hosta (CPU)
    int* host_b = (int*) malloc(sizeof(int) * elements);
    int* host_c = (int*) malloc(sizeof(int) * elements);

    for(int i = 0; i < elements; ++i) { // Wypełnianie tablic fejkowymi danymi
        host_a[i] = i;
        host_b[i] = i;
    }

    // Host

    add_cpu(host_a, host_b, host_c, elements); // Dodawanie na CPU

    for(int i = 0; i < 10; ++i)
        printf("Host %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    memset(host_c, 0, sizeof(int) * elements); // Profilaktyczne zerowanie pamięci - będziemy jeszcze korzystać z tych tablic i lepiej się upewnić, że nie widzimy starego wyniku.

    // GPU

    int* device_a;
    int* device_b;
    int* device_c;
    hipMalloc(&device_a, sizeof(int) * elements); // Allokacja pamięci na urządzeniu (GPU)
    hipMalloc(&device_b, sizeof(int) * elements);
    hipMalloc(&device_c, sizeof(int) * elements);

    hipMemcpy(device_a, host_a, sizeof(int) * elements, hipMemcpyHostToDevice); // Kopiowanie pamięci z hosta na urządzenie - jako ostatni parametr można dać hipMemcpyDefault, wtedy sam się domyśli, w jakim kierunku kopiujemy
    hipMemcpy(device_b, host_b, sizeof(int) * elements, hipMemcpyHostToDevice);

    dim3 block(128);
    dim3 grid(elements / block.x); // Dzielimy tablice wyjściową na bloki po 128 wątków każdy

    add_gpu<<<grid, block>>>(device_a, device_b, device_c); // Dodajemy na GPU
    
    hipMemcpy(host_c, device_c, sizeof(int) * elements, hipMemcpyDeviceToHost); // Sprowadzamy dane z GPU na hosta.
    for(int i = 0; i < 10; ++i)
        printf("GPU 1 %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    memset(host_c, 0, sizeof(int) * elements);

    free(host_a); // Zwalniamy tablice zarezerwowane na hoście, dalej będziemy je allokować jeszcze raz przy pomocy API CUDA
    free(host_b);
    free(host_c);

    // GPU Pinned

    hipHostMalloc(&host_a, sizeof(int) * elements); // Allokujemy pamięć na hoście przy pomocy API CUDA. W przeciwieństwie do tego, co zwraca malloc, ta pamięć nie jest stronicowana - unikamy więc pośredniczącego bufora przy kopiowaniu między hostem a urządzeniem.
    hipHostMalloc(&host_b, sizeof(int) * elements);
    hipHostMalloc(&host_c, sizeof(int) * elements);

    for(int i = 0; i < elements; ++i) {
        host_a[i] = i;
        host_b[i] = i;
    }

    hipMemcpy(device_a, host_a, sizeof(int) * elements, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, sizeof(int) * elements, hipMemcpyHostToDevice);

    add_gpu<<<grid, block>>>(device_a, device_b, device_c);
    
    hipMemcpy(host_c, device_c, sizeof(int) * elements, hipMemcpyDeviceToHost);
    for(int i = 0; i < 10; ++i)
        printf("GPU Pinned %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    memset(host_c, 0, sizeof(int) * elements);

    hipHostFree(host_a); // Pamięć po stronie hosta teraz też musimy zwolnić przy pomocy API CUDA.
    hipHostFree(host_b);
    hipHostFree(host_c);    
    
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    // GPU Mapped

    hipHostAlloc(&host_a, sizeof(int) * elements, hipHostMallocMapped); // Allokujemy po stronie hosta mapowany obszar pamięci. Warning: działa wolno!
    hipHostAlloc(&host_b, sizeof(int) * elements, hipHostMallocMapped);
    hipHostAlloc(&host_c, sizeof(int) * elements, hipHostMallocMapped);

    for(int i = 0; i < elements; ++i) {
        host_a[i] = i;
        host_b[i] = i;
    }

    add_cpu(host_a, host_b, host_c, elements);

    for(int i = 0; i < 10; ++i)
        printf("Host Mapped: %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    memset(host_c, 0, sizeof(int) * elements);
    
    add_gpu<<<grid, block>>>(host_a, host_b, host_c); // Ponieważ używamy mapowanej pamięci, to te same wskaźniki działają dla hosta i GPU.
    hipDeviceSynchronize();

    for(int i = 0; i < 10; ++i)
        printf("GPU Mapped %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);

    hipHostFree(host_a); // Zwalniamy mapowaną pamięć - musimy znowu użyć tej funkcji.
    hipHostFree(host_b);
    hipHostFree(host_c);   

    // GPU Managed

    hipMallocManaged(&host_a, sizeof(int) * elements); // Dla odmiany allokujemy pamięć zarządzaną przez framework CUDA.
    hipMallocManaged(&host_b, sizeof(int) * elements);
    hipMallocManaged(&host_c, sizeof(int) * elements);

    for(int i = 0; i < elements; ++i) {
        host_a[i] = i;
        host_b[i] = i;
    }

    add_cpu(host_a, host_b, host_c, elements);

    for(int i = 0; i < 10; ++i)
        printf("Host Managed: %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    memset(host_c, 0, sizeof(int) * elements);
    
    add_gpu<<<grid, block>>>(host_a, host_b, host_c); // Znowu - te same wskaźniki na hoście i urządzeniu
    hipDeviceSynchronize();

    for(int i = 0; i < 10; ++i)
        printf("GPU Managed %d + %d = %d\n", host_a[i], host_b[i], host_c[i]);

    hipFree(host_a); // Musimy tę pamięć zwolnić przy pomocy hipFree
    hipFree(host_b);
    hipFree(host_c);   

    return 0;
}