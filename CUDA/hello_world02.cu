
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("%d - %d\n", blockIdx.x, threadIdx.x); // blockIdx.x - numer bloku, threadIdx.x - numer wątku w bloku
}

int main() {
    kernel<<<3, 5>>>(); // Trzy bloki po 5 wątków w każdym
    return 0;
}