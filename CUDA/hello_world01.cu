
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("Hello CUDA!\n");
}

int main() {
    kernel<<<1, 32>>>(); // Odpalamy kernel z 1 blokiem i 32 wątkami w tym bloku
    return 0;
}