
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("(%d, %d) - (%d, %d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

int main() {
    dim3 grid(2, 2); // 2D grid z 2x2 blokami
    dim3 block(3, 3); // 3x3 wątki w każdym bloku

    kernel<<<grid, block>>>();
    return 0;
}